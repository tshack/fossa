#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define GPU_MEM_MAX 4000000000


int
main (int argc, char* argv[])
{
    size_t new_size;
    size_t free;
    size_t total;
    size_t alloc_size;
    char* gpu_mem;


    if (argc < 2) {
       fprintf (stderr, "Please specify desired size of GPU memory\n\n"); 
       return 0;
    }
    new_size = atoi (argv[1]);

    hipMemGetInfo (&free, &total);
    free /= 1000000;
    total /= 1000000;

    printf ("GPU Memory: %i/%i MB available\n\n", free, total);

    if (free < new_size) {
        fprintf (stderr, "GPU is already smaller than %i MB\n", new_size);
        exit (0);
    }

    printf ("Shrinking GPU Memory... ");
    fflush (stdout);

    alloc_size = (free - new_size) * sizeof (char) * 1000000;
    hipMalloc ((void**) &gpu_mem, alloc_size);

    printf ("done.\n\n");

    hipMemGetInfo (&free, &total);

    printf ("GPU Memory: %i/%i MB available\n\n", free/1000000, total/1000000);

    printf ("Ctrl-C to exit...\n");

    while (1);
}

