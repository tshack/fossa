#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 50
#define GPU_MEM_MAX 4000000000

// allocate cpu memory and initialite element i to contain the value i
float*
init_cpu_mem (int block_size)
{
    int i;
    float *cpu_mem;
    size_t block_size_t;

    // convert from MB to bytes
    block_size_t = block_size * 250000 * sizeof (float);

    fprintf (stderr, "Initializing a %iMB block of CPU memory... ", block_size);
    cpu_mem = (float*)malloc (block_size_t);

    for (i=0; i<block_size; i++) {
        cpu_mem[i] = (float)i;
    }
    fprintf (stderr, "done.\n\n");

    return cpu_mem;
}


// allocate num_blocks of block_size (MB) GPU memory
float**
init_gpu_mem (int num_blocks, int block_size)
{
    int i;
    float **gpu_mem;

    fprintf (stderr, "Allocating %i %iMB blocks of GPU memory.\n", num_blocks, block_size);

    // convert from MB to bytes
    block_size *= 250000 * sizeof (float);

    gpu_mem = (float**)malloc (sizeof(float*) * num_blocks);

    for (i=0; i<num_blocks; i++) {
        fprintf (stderr, "  Block %02i... ", i);
        hipMalloc ((void **)&gpu_mem[i], block_size);
        if (gpu_mem[i] == NULL) {
            fprintf (stderr, "out of memory.\n");
            fprintf (stderr, "Exiting...\n\n");
            exit (0);
        } else {
            fprintf (stderr, "done.\n");
        }
    }

    return gpu_mem;
}


// free all GPU memory blocks
void
free_gpu_mem (float** gpu_mem, int num_blocks)
{
    int i;

    for (i=0; i<num_blocks; i++) {
        hipFree (gpu_mem[i]);
    }

    free (gpu_mem);
}


// GPU address bus is 32-bit... we can't address more than 4GB
int
sanity_check (int num_blocks, int block_size)
{
    size_t block_size_t;

    block_size_t = block_size * 250000 * sizeof (float);

    if (block_size_t*num_blocks > GPU_MEM_MAX) {
        fprintf (stderr, "Cannot allocate %i %iMB blocks (%1.2fGB).  GPU memory map is limited to %iGB\n",
                num_blocks, block_size, (num_blocks*block_size)/1000.f, GPU_MEM_MAX/1000000000);
        return -1;
    } else {
        return 0;
    }
}


// ripple copy the contents of the CPU memory block through the
// GPU blocks.  check the contents of the final GPU block against the
// CPU block... it's a game of telephone.
int
mem_test (float** gpu_mem, float* cpu_mem, int num_blocks, int block_size)
{
    int i;
    size_t block_size_t;
    float *gpu_tmp;

    block_size_t = block_size * 250000 * sizeof (float);

    fprintf (stderr, "Performing memory test... ");

    // 1st copy CPU block to GPU block 0
    hipMemcpy (gpu_mem[0], cpu_mem, block_size_t, hipMemcpyHostToDevice);

    // now ripple the CPU block information through the GPU memory blocks
    for (i=1; i<num_blocks; i++) {
        hipMemcpy (gpu_mem[i], gpu_mem[i-1], block_size_t, hipMemcpyDeviceToDevice);
    }

    gpu_tmp = (float*)malloc (block_size_t);

    hipMemcpy (gpu_tmp, gpu_mem[num_blocks-1], block_size_t, hipMemcpyDeviceToHost);

    // now compare the CPU block to the last GPU block
    for (i=0; i<block_size; i++) {
        if (gpu_tmp[i] != cpu_mem[i]) {
            fprintf (stderr, "FAILED\n\n");
            free (gpu_tmp);
            return -1;
        }
    }
    fprintf (stderr, "PASSED\n\n");
    free (gpu_tmp);

    return 0;
}


int
main (int argc, char* argv[])
{
    int i;
    int num_blocks;
    int result;
    float  *cpu_mem;
    float **gpu_mem;

    if (argc < 2) {
       fprintf (stderr, "Please specify # of 50MB Blocks to allocate\n\n"); 
       return 0;
    }
    num_blocks = atoi (argv[1]);

    if (sanity_check (num_blocks, BLOCK_SIZE) == -1) {
        return 0;
    }

    cpu_mem = init_cpu_mem (BLOCK_SIZE);
    gpu_mem = init_gpu_mem (num_blocks, BLOCK_SIZE);
   

    mem_test (gpu_mem, cpu_mem, num_blocks, BLOCK_SIZE);


    free_gpu_mem (gpu_mem, num_blocks);

    return 0;
}

